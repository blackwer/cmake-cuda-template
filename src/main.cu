#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

std::ostream &operator<<(std::ostream &out, hipDeviceProp_t const &props) {
    // https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaDeviceProp.html
    out << "Device name: " << props.name << std::endl;
    out << "Multiprocessor count: " << props.multiProcessorCount << std::endl;
    out << "Clock rate (GHz): " << props.clockRate / 1E6 << std::endl;
    out << "Concurrent kernel execution: " << (props.concurrentKernels ? "yes" : "no") << std::endl;
    out << "Compute capability: " << props.major << props.minor << std::endl;
    out << "Max threads dim: [" << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", "
        << props.maxThreadsDim[2] << "]\n";
    out << "Max threads/block: " << props.maxThreadsPerBlock << std::endl;
    out << "Max threads per multiprocessor: " << props.maxThreadsPerMultiProcessor << std::endl;
    out << "Warp size (threads): " << props.warpSize << std::endl;
    out << "Global memory (GiB): " << (double)props.totalGlobalMem / (1024 * 1024 * 1024) << std::endl;
    out << "Global memory peak clock rate (GHz): " << props.memoryClockRate / 1E6 << std::endl;
    out << "Global Memory bus width (bytes): " << props.memoryBusWidth / 8 << std::endl;
    // (DDR = factor of 2), (mem clock in kHz = factor of 1000), (bus width in bits = factor of 1/8/1024^3),
    out << "Theoretical memory throughput (GiB/s): "
        << (double)2 * 1000 * props.memoryClockRate * props.memoryBusWidth / 8 / (1024 * 1024 * 1024) << std::endl;
    out << "Shared memory/block (KiB): " << props.sharedMemPerBlock / 1024 << std::endl;
    out << "Async engine count: " << props.asyncEngineCount << std::endl;
    out << "Global L1 cache support: " << (props.globalL1CacheSupported ? "yes" : "no") << std::endl;
    out << "Local L1 cache support: " << (props.localL1CacheSupported ? "yes" : "no") << std::endl;
    out << "L2 cache size (MiB): " << props.l2CacheSize / 1024 << std::endl;

    return out;
}

int main(int argc, char *argv[]) {
    int n_devices;
    hipGetDeviceCount(&n_devices);

    if (!n_devices) {
        std::cerr << "No cuda devices found\n";
        return 1;
    }

    int device_id;
    hipGetDevice(&device_id);

    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, device_id);
    std::cout << device_props;

    return 0;
}
